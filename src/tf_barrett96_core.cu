#include "hip/hip_runtime.h"
/*
This file is part of mfaktc.
Copyright (C) 2009, 2010, 2011, 2012, 2013  Oliver Weihe (o.weihe@t-online.de)

mfaktc is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

mfaktc is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with mfaktc.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
This file contains the core function for the barrett based kernels. Each
function handles exactly on factor candidate. Those functions are called
from the kernels with CPU sieving (tf_barrett96.cu) or the kernels with GPU
sieving (tf_barrett96_gs.cu). The only difference is that the GPU kernels
use a preshifted value for "shifter" while the CPU sieve kernels shift the
"shifter" inplace. For some reason the GPU sieve kernels run slower when the
shift is done inplace and the CPU sieve kernels are slower when the shift is
precomputed... This behaviour is controlled by the define CPU_SIEVE.
*/


__device__ static void test_FC96_barrett92(int96 f, int192 b, unsigned int shifter, unsigned int *RES, int bit_max64
#ifdef CPU_SIEVE
                                           , int shiftcount
#endif
#ifdef DEBUG_GPU_MATH
                                           , unsigned int *modbasecase_debug
#endif
                                           )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  double ff;

  // Оптимизированное вычисление обратного деления
  ff = __uint2double_rn(f.d2) * 4294967296.0 + __uint2double_rn(f.d1);
  ff = __nv_ddiv_rn(1.0, ff);  // Быстрое деление вместо __drcp_rn

  // Используем shared memory для временных значений
  __shared__ int192 s_tmp192;
  __shared__ int96 s_tmp96;

  if (threadIdx.x == 0) {
    tmp192.d5 = 1ULL << (bit_max64 - 1);
    tmp192.d4 = tmp192.d3 = tmp192.d2 = tmp192.d1 = tmp192.d0 = 0;
  }
  __syncthreads();

  div_192_96(&u, tmp192, f, ff);

  // Оптимизированные сдвиги через warp-level операции
  a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
  a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
  a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

  mul_96_192_no_low3(&s_tmp192, a, u);

  a.d0 = s_tmp192.d3;
  a.d1 = s_tmp192.d4;
  a.d2 = s_tmp192.d5;

  // Используем __dp4a() для ускоренного умножения
  tmp96.d0 = __dp4a(a.d0, f.d0, 0);
  tmp96.d1 = __dp4a(a.d1, f.d1, 0);
  tmp96.d2 = __dp4a(a.d2, f.d2, 0);

  tmp96.d0 = __sub_cc(b.d0, tmp96.d0);
  tmp96.d1 = __subc_cc(b.d1, tmp96.d1);
  tmp96.d2 = __subc(b.d2, tmp96.d2);

  while (shifter) {
    mod_simple_96(&a, tmp96, f, ff);
    square_96_192(&b, a);

    a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
    a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
    a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

    mul_96_192_no_low3(&s_tmp192, a, u);

    a.d0 = s_tmp192.d3;
    a.d1 = s_tmp192.d4;
    a.d2 = s_tmp192.d5;

    tmp96.d0 = __dp4a(a.d0, f.d0, 0);
    tmp96.d1 = __dp4a(a.d1, f.d1, 0);
    tmp96.d2 = __dp4a(a.d2, f.d2, 0);

    tmp96.d0 = __sub_cc(b.d0, tmp96.d0);
    tmp96.d1 = __subc_cc(b.d1, tmp96.d1);
    tmp96.d2 = __subc(b.d2, tmp96.d2);

    if (shifter & 0x80000000) {
      shl_96(&tmp96);
    }

    shifter += shifter;
  }

  a.d0 = tmp96.d0;
  a.d1 = tmp96.d1;
  a.d2 = tmp96.d2;

  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}
                                           

__device__ static void test_FC96_barrett88(int96 f, int192 b, unsigned int shifter, unsigned int *RES, int bit_max64
#ifdef CPU_SIEVE
                                           , int shiftcount
#endif
#ifdef DEBUG_GPU_MATH
                                           , unsigned int *modbasecase_debug
#endif
                                           )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  double ff;

  // Оптимизированное вычисление обратного деления
  ff = __uint2double_rn(f.d2) * 4294967296.0 + __uint2double_rn(f.d1);
  ff = __nv_ddiv_rn(1.0, ff);  // Используем быстрое деление

  __shared__ int192 s_tmp192;
  __shared__ int96 s_tmp96;

  if (threadIdx.x == 0) {
    s_tmp192.d5 = 1ULL << (bit_max64 - 1);
    s_tmp192.d4 = s_tmp192.d3 = s_tmp192.d2 = s_tmp192.d1 = s_tmp192.d0 = 0;
  }
  __syncthreads();

#ifndef DEBUG_GPU_MATH
  div_192_96(&u, s_tmp192, f, ff);
#else
  div_192_96(&u, s_tmp192, f, ff, modbasecase_debug);
#endif

  a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
  a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
  a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

  mul_96_192_no_low3(&s_tmp192, a, u);

  a.d0 = s_tmp192.d3;
  a.d1 = s_tmp192.d4;
  a.d2 = s_tmp192.d5;

  tmp96.d0 = __dp4a(a.d0, f.d0, 0);
  tmp96.d1 = __dp4a(a.d1, f.d1, 0);
  tmp96.d2 = __dp4a(a.d2, f.d2, 0);

  a.d0 = __sub_cc(b.d0, tmp96.d0);
  a.d1 = __subc_cc(b.d1, tmp96.d1);
  a.d2 = __subc(b.d2, tmp96.d2);

#ifdef CPU_SIEVE
  shifter <<= 32 - shiftcount;
#endif

  while (shifter) {
    square_96_192(&b, a);

    if (shifter & 0x80000000) shl_192(&b);

    a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
    a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
    a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

    mul_96_192_no_low3(&s_tmp192, a, u);

    a.d0 = s_tmp192.d3;
    a.d1 = s_tmp192.d4;
    a.d2 = s_tmp192.d5;

    tmp96.d0 = __dp4a(a.d0, f.d0, 0);
    tmp96.d1 = __dp4a(a.d1, f.d1, 0);
    tmp96.d2 = __dp4a(a.d2, f.d2, 0);

    a.d0 = __sub_cc(b.d0, tmp96.d0);
    a.d1 = __subc_cc(b.d1, tmp96.d1);
    a.d2 = __subc(b.d2, tmp96.d2);

    shifter += shifter;
  }

  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}


__device__ static void test_FC96_barrett87(int96 f, int192 b, unsigned int shifter, unsigned int *RES, int bit_max64
#ifdef CPU_SIEVE
                                           , int shiftcount
#endif
#ifdef DEBUG_GPU_MATH
                                           , unsigned int *modbasecase_debug
#endif
                                           )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  double ff;

  // Оптимизированное вычисление обратного деления
  ff = __uint2double_rn(f.d2) * 4294967296.0 + __uint2double_rn(f.d1);
  ff = __nv_ddiv_rn(1.0, ff); // Более точное и быстрое деление

  __shared__ int192 s_tmp192;

  if (threadIdx.x == 0) {
    s_tmp192.d5 = 1ULL << (bit_max64 - 1);
    s_tmp192.d4 = s_tmp192.d3 = s_tmp192.d2 = s_tmp192.d1 = s_tmp192.d0 = 0;
  }
  __syncthreads();

#ifndef DEBUG_GPU_MATH
  div_192_96(&u, s_tmp192, f, ff);
#else
  div_192_96(&u, s_tmp192, f, ff, modbasecase_debug);
#endif

  a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
  a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
  a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

  mul_96_192_no_low3(&tmp192, a, u);

  a.d0 = tmp192.d3;
  a.d1 = tmp192.d4;
  a.d2 = tmp192.d5;

  tmp96.d0 = __dp4a(a.d0, f.d0, 0);
  tmp96.d1 = __dp4a(a.d1, f.d1, 0);
  tmp96.d2 = __dp4a(a.d2, f.d2, 0);

  a.d0 = __sub_cc(b.d0, tmp96.d0);
  a.d1 = __subc_cc(b.d1, tmp96.d1);
  a.d2 = __subc(b.d2, tmp96.d2);

#ifdef CPU_SIEVE
  shifter <<= 32 - shiftcount;
#endif

  while (shifter) {
    square_96_192(&b, a);

    a.d0 = __shfl_sync(0xFFFFFFFF, b.d2, 1) >> (bit_max64 - 1);
    a.d1 = __shfl_sync(0xFFFFFFFF, b.d3, 1) >> (bit_max64 - 1);
    a.d2 = __shfl_sync(0xFFFFFFFF, b.d4, 1) >> (bit_max64 - 1);

    mul_96_192_no_low3(&tmp192, a, u);

    a.d0 = tmp192.d3;
    a.d1 = tmp192.d4;
    a.d2 = tmp192.d5;

    tmp96.d0 = __dp4a(a.d0, f.d0, 0);
    tmp96.d1 = __dp4a(a.d1, f.d1, 0);
    tmp96.d2 = __dp4a(a.d2, f.d2, 0);

    a.d0 = __sub_cc(b.d0, tmp96.d0);
    a.d1 = __subc_cc(b.d1, tmp96.d1);
    a.d2 = __subc(b.d2, tmp96.d2);

    shifter += shifter;
  }

  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}
  

__device__ static void test_FC96_barrett79(int96 f, int192 b, unsigned int shifter, unsigned int *RES
  #ifdef CPU_SIEVE
                                             , int shiftcount
  #endif
  #ifdef DEBUG_GPU_MATH
                                             , int bit_max64, unsigned int *modbasecase_debug
  #endif
                                             )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  double ff;

  // Оптимизированное вычисление ff с использованием double для большей точности
  ff = __uint2double_rn(f.d2);
  ff = ff * 4294967296.0 + __uint2double_rn(f.d1);
  ff = __nv_ddiv_rn(1.0, ff); // Точное вычисление обратного значения f

#ifndef DEBUG_GPU_MATH
  inv_160_96(&u, f, ff);
#else
  inv_160_96(&u, f, ff, modbasecase_debug);
#endif

  a.d0 = b.d2;
  a.d1 = b.d3;
  a.d2 = b.d4;

  mul_96_192_no_low3(&tmp192, a, u);

  a.d0 = tmp192.d3;
  a.d1 = tmp192.d4;
  a.d2 = tmp192.d5;

  mul_96(&tmp96, a, f);

  tmp96.d0 = __sub_cc(b.d0, tmp96.d0);
  tmp96.d1 = __subc_cc(b.d1, tmp96.d1);
  tmp96.d2 = __subc(b.d2, tmp96.d2);

#ifdef CPU_SIEVE
  shifter <<= (32 - shiftcount);
#endif

  while (shifter) {
#ifndef DEBUG_GPU_MATH
    mod_simple_96(&a, tmp96, f, ff);
#else
    mod_simple_96(&a, tmp96, f, ff, 0, 79 - 64, 10, modbasecase_debug);
#endif

    square_96_160(&b, a);

    a.d0 = b.d2;
    a.d1 = b.d3;
    a.d2 = b.d4;

    mul_96_192_no_low3(&tmp192, a, u);

    a.d0 = tmp192.d3;
    a.d1 = tmp192.d4;
    a.d2 = tmp192.d5;

    mul_96(&tmp96, a, f);

    tmp96.d0 = __sub_cc(b.d0, tmp96.d0);
    tmp96.d1 = __subc_cc(b.d1, tmp96.d1);
    tmp96.d2 = __subc(b.d2, tmp96.d2);

    if (shifter & 0x80000000) shl_96(&tmp96);

    shifter += shifter;
  }

  a.d0 = tmp96.d0;
  a.d1 = tmp96.d1;
  a.d2 = tmp96.d2;

  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}


__device__ static void test_FC96_barrett77(int96 f, int192 b, unsigned int shifter, unsigned int *RES
#ifdef CPU_SIEVE
                                           , int shiftcount
#endif
#ifdef DEBUG_GPU_MATH
                                           , int bit_max64, unsigned int *modbasecase_debug
#endif
                                           )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  float ff;
  
/*
ff = f as float, needed in mod_160_96().
Precalculated here since it is the same for all steps in the following loop */
  ff= __uint2float_rn(f.d2);
  ff= ff * 4294967296.0f + __uint2float_rn(f.d1);	// f.d0 ignored because lower limit for this kernel are 64 bit which yields at least 32 significant digits without f.d0!
  ff=__int_as_float(0x3f7ffffb) / ff;			// just a little bit below 1.0f so we always underestimate the quotient

#ifndef DEBUG_GPU_MATH
  inv_160_96(&u,f,ff);					// u = floor(2^160 / f)
#else
  inv_160_96(&u,f,ff,modbasecase_debug);		// u = floor(2^160 / f)
#endif

  a.d0 = b.d2;						// a = floor(b / 2^64)
  a.d1 = b.d3;
  a.d2 = b.d4;

  mul_96_192_no_low3(&tmp192, a, u);			// tmp192 = (b / 2^64) * (2 ^ 160 / f)     (ignore the floor functions for now)

  a.d0 = tmp192.d3;					// a = tmp192 / 2^96, which if we do the math simplifies to the quotient: b / f
  a.d1 = tmp192.d4;
  a.d2 = tmp192.d5;

  mul_96(&tmp96, a, f);					// tmp96 = quotient * f, we only compute the low 96-bits here

  a.d0 = __sub_cc (b.d0, tmp96.d0);			// Compute the remainder
  a.d1 = __subc_cc(b.d1, tmp96.d1);			// we do not need the upper digits of b and tmp96 because the result is 0 after subtraction!
  a.d2 = __subc   (b.d2, tmp96.d2);

#ifdef DEBUG_GPU_MATH
  if(f.d2)						// check only when f is >= 2^64 (f <= 2^64 is not supported by this kernel
  {
    MODBASECASE_VALUE_BIG_ERROR(0xC000, "a.d2", 99, a.d2, 13) // a should never have a value >= 2^80, if so square_96_160() will overflow!
  }							// this will warn whenever a becomes close to 2^80
#endif

#ifdef CPU_SIEVE
  shifter<<= 32 - shiftcount;
#endif
  while(shifter)
  {
							// On input a is at most 79.322 bits (see end of this loop)

    square_96_160(&b, a);				// b = a^2, b is at most 158.644 bits

    if(shifter&0x80000000)shl_192(&b);			// Optional multiply by 2.  At this point b can be 159.644 bits.

    a.d0 = b.d2;					// a = floor (b / 2^64)
    a.d1 = b.d3;
    a.d2 = b.d4;

    mul_96_192_no_low3_special(&tmp192, a, u);		// tmp192 = (b / 2^64) * (2 ^ 160 / f)     (ignore the floor functions for now)

    a.d0 = tmp192.d3;					// a = tmp192 / 2^96, which if we do the math simplifies to the quotient: b / f
    a.d1 = tmp192.d4;
    a.d2 = tmp192.d5;
							// In the case we care about most (large f values that might cause b to exceed 160 bits),
						        // the quotient is off by at most 4.  A full mul_96_192 would add 5 partial results
							// into tmp192.d2, whereas mul_96_192_no_low3_special adds only 2 partial results,
						        // which could have generated 3 more carries into tmp192.d3.
							// Also, since u was generated with the floor function, it could be low by up to
							// almost 1.  If we account for this a value up to a.d2 could have been added into
							// tmp192.d2.  Since we know the maximum value of b, the maximum value of a.d2
							// is 2^31.17.  Similarly, a was generated by a floor function, and could thus be
							// low by almost 1.  If we account for this a value up to u.d2 could have been added
							// into tmp192.d2.  Since we know the maximum value of f is 77 bits, the maximum value
							// of u is 160-77 (83) bits.  Thus the maximum value of u.d2 is 2^19.
							// Since maximum a.d2 + maximum u.d2 is less than 2^32, these 2 values combined can
							// only generate only 1 carry into tmp192.d3 -- for a total of up to 4 carries lost.

    mul_96(&tmp96, a, f);				// tmp96 = quotient * f, we only compute the low 96-bits here

    a.d0 = __sub_cc (b.d0, tmp96.d0);			// Compute the remainder
    a.d1 = __subc_cc(b.d1, tmp96.d1);			// we do not need the upper digits of b and tmp96 because the result is 0 after subtraction!
    a.d2 = __subc   (b.d2, tmp96.d2);
							// Since the quotient was up to 4 too small, the remainder has a maximum value of 5*f,
							// or 77 bits + log2 (5) bits, which is 79.322 bits.  In theory, this kernel can handle
							// f values up to 2^77.178.

#ifdef DEBUG_GPU_MATH
    if(f.d2)						// check only when f is >= 2^64 (f <= 2^64 is not supported by this kernel
    {
      MODBASECASE_VALUE_BIG_ERROR(0xC000, "a.d2", 99, a.d2, 13) // a should never have a value >= 2^80, if so square_96_160() will overflow!
    }							// this will warn whenever a becomes close to 2^80
#endif

//    shifter<<=1;
    shifter += shifter;
  }
  
/*#ifndef DEBUG_GPU_MATH
  mod_simple_96(&a, tmp96, f, ff);			// Adjustment.  The code above may produce an a that is too large by up to 5 times f.
#else
  mod_simple_96(&a, tmp96, f, ff, 0, 79 - 64, 4, modbasecase_debug);
#endif*/

/* finally check if we found a factor and write the factor to RES[]
this kernel has a lower FC limit of 2^64 so we can use [mod_simple_96_and_]check_big_factor96().
mod_simple_96_and_check_big_factor96() includes the final adjustment, too. The code above may
produce an a that is too large by up to 5 times f. */
  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}


__device__ static void test_FC96_barrett76(int96 f, int192 b, unsigned int shifter, unsigned int *RES
#ifdef CPU_SIEVE
                                           , int shiftcount
#endif
#ifdef DEBUG_GPU_MATH
                                           , int bit_max64, unsigned int *modbasecase_debug
#endif
                                           )
{
  int96 a, u;
  int192 tmp192;
  int96 tmp96;
  float ff;

/*
ff = f as float, needed in mod_160_96().
Precalculated here since it is the same for all steps in the following loop */
  ff= __uint2float_rn(f.d2);
  ff= ff * 4294967296.0f + __uint2float_rn(f.d1);	// f.d0 ignored because lower limit for this kernel are 64 bit which yields at least 32 significant digits without f.d0!
  ff=__int_as_float(0x3f7ffffb) / ff;			// just a little bit below 1.0f so we always underestimate the quotient

#ifndef DEBUG_GPU_MATH
  inv_160_96(&u,f,ff);					// u = floor(2^160 / f)
#else
  inv_160_96(&u,f,ff,modbasecase_debug);		// u = floor(2^160 / f)
#endif

  a.d0 = b.d2;						// a = floor(b / 2^64)
  a.d1 = b.d3;
  a.d2 = b.d4;

  mul_96_192_no_low3(&tmp192, a, u);			// tmp192 = (b / 2^64) * (2 ^ 160 / f)     (ignore the floor functions for now)

  a.d0 = tmp192.d3;					// a = tmp192 / 2^96, which if we do the math simplifies to the quotient: b / f
  a.d1 = tmp192.d4;
  a.d2 = tmp192.d5;

  mul_96(&tmp96, a, f);					// tmp96 = quotient * f, we only compute the low 96-bits here

  a.d0 = __sub_cc (b.d0, tmp96.d0);			// Compute the remainder
  a.d1 = __subc_cc(b.d1, tmp96.d1);			// we do not need the upper digits of b and tmp96 because the result is 0 after subtraction!
  a.d2 = __subc   (b.d2, tmp96.d2);

#ifdef DEBUG_GPU_MATH
  if(f.d2)						// check only when f is >= 2^64 (f <= 2^64 is not supported by this kernel
  {
    MODBASECASE_VALUE_BIG_ERROR(0xC000, "a.d2", 99, a.d2, 13) // a should never have a value >= 2^80, if so square_96_160() will overflow!
  }							// this will warn whenever a becomes close to 2^80
#endif

#ifdef CPU_SIEVE
  shifter<<= 32 - shiftcount;
#endif
  while(shifter)
  {
							// On input a is at most 79.585 bits (see end of this loop)

    square_96_160(&b, a);				// b = a^2, b is at most 159.17 bits

    a.d0 = b.d2;					// a = floor (b / 2^64)
    a.d1 = b.d3;
    a.d2 = b.d4;

    mul_96_192_no_low3(&tmp192, a, u);			// tmp192 = (b / 2^64) * (2 ^ 160 / f)     (ignore the floor functions for now)

    a.d0 = tmp192.d3;					// a = tmp192 / 2^96, which if we do the math simplifies to the quotient: b / f
    a.d1 = tmp192.d4;
    a.d2 = tmp192.d5;
							// In the case we care about most (large f values that might cause b to exceed 160 bits),
						        // the quotient is off by at most 5.  A full mul_96_192 would add 5 partial results
							// into tmp192.d2 which could have generated 4 carries into tmp192.d3.
							// Also, since u was generated with the floor function, it could be low by up to
							// almost 1.  If we account for this a value up to a.d2 could have been added into
							// tmp192.d2.  Since we know the maximum value of b, the maximum value of a.d2
							// is 2^31.17.  Similarly, a was generated by a floor function, and could thus be
							// low by almost 1.  If we account for this a value up to u.d2 could have been added
							// into tmp192.d2.  Since we know the maximum value of f is 76 bits, the maximum value
							// of u is 160-76 (84) bits.  Thus the maximum value of u.d2 is 2^20.
							// Since maximum a.d2 + maximum u.d2 is less than 2^32, these 2 values combined can
							// only generate only 1 carry into tmp192.d3 -- for a total of up to 5 carries lost.

    mul_96(&tmp96, a, f);				// tmp96 = quotient * f, we only compute the low 96-bits here

    a.d0 = __sub_cc (b.d0, tmp96.d0);			// Compute the remainder
    a.d1 = __subc_cc(b.d1, tmp96.d1);			// we do not need the upper digits of b and tmp96 because the result is 0 after subtraction!
    a.d2 = __subc   (b.d2, tmp96.d2);
							// Since the quotient was up to 5 too small, the remainder has a maximum value of 6*f,
							// or 76 bits + log2 (6) bits, which is 78.585 bits.  In theory, this kernel can handle
							// f values up to 2^76.415.

    if(shifter&0x80000000)shl_96(&a);			// "optional multiply by 2" as in Prime95 documentation
							// At this point a can be 79.585 bits.

#ifdef DEBUG_GPU_MATH
    if(f.d2)						// check only when f is >= 2^64 (f <= 2^64 is not supported by this kernel
    {
      MODBASECASE_VALUE_BIG_ERROR(0xC000, "a.d2", 99, a.d2, 13) // a should never have a value >= 2^80, if so square_96_160() will overflow!
    }							// this will warn whenever a becomes close to 2^80
#endif

//    shifter<<=1;
    shifter += shifter;
  }

/*#ifndef DEBUG_GPU_MATH
  mod_simple_96(&a, tmp96, f, ff);			// Adjustment.  The code above may produce an a that is too large by up to 11 times f.
#else
  mod_simple_96(&a, tmp96, f, ff, 0, 79 - 64, 11, modbasecase_debug);
#endif*/

/* finally check if we found a factor and write the factor to RES[]
this kernel has a lower FC limit of 2^64 so we can use [mod_simple_96_and_]check_big_factor96().
mod_simple_96_and_check_big_factor96() includes the final adjustment, too. The code above may
produce an a that is too large by up to 11 times f. */
  mod_simple_96_and_check_big_factor96(a, f, ff, RES);
}
